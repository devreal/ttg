#include "hip/hip_runtime.h"

#include <Kokkos_Core.hpp>

//#include "ttg/device/device.h"

#include <iostream>


// fwd-decl
namespace ttg::device {
hipStream_t current_stream();
} // namespace ttg::device

//#define KOKKOS_PLGSY_HOST

#ifdef KOKKOS_PLGSY_HOST
#define KOKKOS_SPACE Kokkos::HostSpace
#define KOKKOS_POLICY Kokkos::Serial
#define KOKKOS_POLICY_INSTANCE Kokkos::Serial()
#else
#define KOKKOS_SPACE Kokkos::Cuda::memory_space
#define KOKKOS_POLICY Kokkos::Cuda
#define KOKKOS_POLICY_INSTANCE Kokkos::Cuda(ttg::device::current_stream())
#endif

#include "random.h"

namespace detail {

  template<typename T>
  struct num_elems : std::integral_constant<int, 1>
  { };

  template<typename T>
  struct num_elems<std::complex<T>> : std::integral_constant<int, 2>
  { };

  template<typename T>
  static constexpr int num_elems_v = num_elems<T>::value;

  template<typename T>
  struct is_complex : std::integral_constant<bool, false>
  { };

  template<typename T>
  struct is_complex<std::complex<T>> : std::integral_constant<bool, true>
  { };

  template<typename T>
  static constexpr bool is_complex_v = is_complex<T>::value;
} // namespace detail

template <typename T>
void CORE_plgsy_device( T bump, int m, int n, T *A, int lda,
                        int gM, int m0, int n0, unsigned long long int seed ) {
  static constexpr int nbelem = detail::num_elems_v<T>;
  auto layout = Kokkos::LayoutStride(m, lda, n, 1);
  auto view = Kokkos::View<T**, Kokkos::LayoutStride,
                           KOKKOS_SPACE>(A, layout);

  //std::cout << "CORE_plgsy_device stream " << ttg::device::current_stream()
  //          << " buffer " << A
  //          << " m " << m << " n " << n
  //          << " lda " << lda << " gM " << gM << " m0 " << m0
  //          << " n0 " << n0 << " seed " << seed << std::endl;

  auto rnd = KOKKOS_LAMBDA(unsigned long long int n, unsigned long long int seed) {
    unsigned long long int a_k, c_k, ran;
    a_k = Rnd64_A;
    c_k = Rnd64_C;
    ran = seed;
    for (int i = 0; n; n >>= 1, ++i) {
      if (n & 1)
        ran = a_k * ran + c_k;
      c_k *= (a_k + 1);
      a_k *= a_k;
    }

    return ran;
  };

  auto gen = KOKKOS_LAMBDA(unsigned long long ran){
    if constexpr(detail::is_complex_v<T>) {
      return T((0.5f - ran * RndF_Mul), (0.5f - ran * RndF_Mul));
    } else {
      return (0.5f - ran * RndF_Mul);
    }
  };


  KOKKOS_POLICY pol = KOKKOS_POLICY_INSTANCE;

  if ( m0 == n0 ) {
    /* diagonal */
    Kokkos::parallel_for("diagonal",
      Kokkos::MDRangePolicy<KOKKOS_POLICY, Kokkos::Rank<2>>(pol, {0, 0}, {n, m}),
      KOKKOS_LAMBDA(int row, int col) {
        unsigned long long int jump = (unsigned long long int)m0 + (unsigned long long int)n0 * (unsigned long long int)gM;
        jump += std::min(col, row)*(gM+1);
        unsigned long long int ran;
        ran = rnd( nbelem * jump, seed );
        for (int i = 0; i < row; ++i) {
          for (int j = i; j < col; ++j) {
            ran = Rnd64_A*ran + Rnd64_C;
          }
        }
        view(row, col) = gen(ran);
        if (row == col) {
          /* bump diagonal */
          view(row, col) += bump;
        }
      }
    );
  } else if (m0 > n0) {
    /* Lower part */
    Kokkos::parallel_for("lower part",
      Kokkos::MDRangePolicy<KOKKOS_POLICY, Kokkos::Rank<2>>(pol, {0, 0}, {n, m}),
      KOKKOS_LAMBDA(int row, int col) {
        unsigned long long int jump = (unsigned long long int)m0 + (unsigned long long int)n0 * (unsigned long long int)gM;
        jump += row*gM;
        unsigned long long int ran;
        ran = rnd( nbelem * jump, seed );
        for (int i = 0; i < row*n+col; ++i) {
            ran = Rnd64_A*ran + Rnd64_C;
        }
        view(row, col) = gen(ran);
      }
    );
  } else {
    /* upper part */
    Kokkos::parallel_for("upper part",
      Kokkos::MDRangePolicy<KOKKOS_POLICY, Kokkos::Rank<2>>(pol, {0, 0}, {n, m}),
      KOKKOS_LAMBDA(int row, int col) {
        unsigned long long int jump = (unsigned long long int)m0 + (unsigned long long int)n0 * (unsigned long long int)gM;
        jump += col*gM;
        unsigned long long int ran;
        ran = rnd( nbelem * jump, seed );
        for (int i = 0; i < col*n+row; ++i) {
            ran = Rnd64_A*ran + Rnd64_C;
        }
        view(row, col) = gen(ran);
      }
    );
  }
}

/* implicit instantiations */
template void CORE_plgsy_device<float>(float bump, int m, int n, float *A, int lda,
                        int gM, int m0, int n0, unsigned long long int seed);

template void CORE_plgsy_device<double>(double bump, int m, int n, double *A, int lda,
                        int gM, int m0, int n0, unsigned long long int seed);

template void CORE_plgsy_device<std::complex<float>>(std::complex<float> bump, int m, int n, std::complex<float> *A, int lda,
                        int gM, int m0, int n0, unsigned long long int seed);

template void CORE_plgsy_device<std::complex<double>>(std::complex<double> bump, int m, int n, std::complex<double> *A, int lda,
                        int gM, int m0, int n0, unsigned long long int seed);


void kokkos_init(int& argc, char* argv[]) {
  Kokkos::initialize(argc, argv);
}

void kokkos_finalize() {
  Kokkos::finalize();
}